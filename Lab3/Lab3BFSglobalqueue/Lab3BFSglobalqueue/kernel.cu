
#include "hip/hip_runtime.h"

#include "hip/device_functions.h"


#include <stdio.h>
#include <stdlib.h>

//definition of the logic gates
#define AND     0
#define OR      1
#define NAND    2
#define NOR     3
#define XOR     4
#define XNOR    5

int read_input_one_two_four(int** input1, char* filepath) {
	FILE* fp = fopen(filepath, "r");
	if (fp == NULL) {
		fprintf(stderr, "Couldn't open file for reading\n");
		exit(1);
	}

	int counter = 0;
	int len;
	int length = fscanf(fp, "%d", &len);
	*input1 = (int*)malloc(len * sizeof(int));

	int temp1;

	while (fscanf(fp, "%d", &temp1) == 1) {
		(*input1)[counter] = temp1;

		counter++;
	}

	fclose(fp);
	return len;




}
int read_input_three(int** input1, int** input2, int** input3, int** input4, char* filepath) {
	FILE* fp = fopen(filepath, "r");
	if (fp == NULL) {
		fprintf(stderr, "Couldn't open file for reading\n");
		exit(1);
	}

	int counter = 0;
	int len;
	int length = fscanf(fp, "%d", &len);
	*input1 = (int*)malloc(len * sizeof(int));
	*input2 = (int*)malloc(len * sizeof(int));
	*input3 = (int*)malloc(len * sizeof(int));
	*input4 = (int*)malloc(len * sizeof(int));



	int temp1;
	int temp2;
	int temp3;
	int temp4;
	while (fscanf(fp, "%d,%d,%d,%d", &temp1, &temp2, &temp3, &temp4) == 4) {
		(*input1)[counter] = temp1;
		(*input2)[counter] = temp2;
		(*input3)[counter] = temp3;
		(*input4)[counter] = temp4;
		counter++;
	}

	fclose(fp);
	return len;

}

__device__ int gate_solver(int gate, int input1, int input2)
{
	int bit;
	switch (gate) {
	case AND:
		bit = input1 & input2;
		break;
	case OR:
		bit = input1 | input2;
		break;
	case NAND:
		bit = !(input1 & input2);
		break;
	case NOR:
		bit = !(input1 | input2);
		break;
	case XOR:
		if ((input1 == 0 && input2 == 1) || (input1 == 1 && input2 == 0)) {
			bit = 1;
		}
		else {
			bit = 0;
		}
		break;
	case XNOR:
		if ((input1 == 0 && input2 == 0) || (input1 == 1 && input2 == 1)) {
			bit = 1;
		}
		else {
			bit = 0;
		}
		break;
	}
	return bit;
}

__global__ void global_queue(int* currLevelNodes_h, int *nodePtrs_h, int *nodeNeighbors_h, int *nodeVisited_h, int *nodeOutput_h, int *nodeGate_h, int *nodeInput_h, int *nextLevelNodes_h, int *numNextLevelNodes_h, int numCurrLevelNodes)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	while(index<numCurrLevelNodes){
		int node = currLevelNodes_h[index];
		for (int j = nodePtrs_h[node]; j < nodePtrs_h[node + 1]; j++) {
			int neighbor = nodeNeighbors_h[j];
			if (!nodeVisited_h[neighbor]) {
				nodeVisited_h[neighbor] = 1;
				nodeOutput_h[neighbor] = gate_solver(nodeGate_h[neighbor], nodeOutput_h[node], nodeInput_h[neighbor]);
				atomicExch(&nextLevelNodes_h[atomicAdd(numNextLevelNodes_h, 1)], neighbor);
			}
		}
		index += stride;

	}

}



int main() {
	FILE* nextlevelnodes_file, * nodeoutput_file;
	nextlevelnodes_file = fopen("nextLevelNodes.txt", "w");
	nodeoutput_file = fopen("nodeOutput.txt", "w");
	// Variables
	int numNodePtrs;
	int numNodes;
	int* nodePtrs_h;
	int* nodeNeighbors_h;
	int* nodeVisited_h;
	int numTotalNeighbors_h;
	int* currLevelNodes_h;
	int numCurrLevelNodes;
	int* numNextLevelNodes_h;
	int* nodeGate_h;
	int* nodeInput_h;
	int* nodeOutput_h;



	//output
	int* nextLevelNodes_h = (int*)malloc(sizeof(int) * 100000);




	numNodePtrs = read_input_one_two_four(&nodePtrs_h, "input1.raw");

	numTotalNeighbors_h = read_input_one_two_four(&nodeNeighbors_h, "input2.raw");

	numNodes = read_input_three(&nodeVisited_h, &nodeGate_h, &nodeInput_h, &nodeOutput_h, "input3.raw");

	numCurrLevelNodes = read_input_one_two_four(&currLevelNodes_h, "input4.raw");

	//cuda variables
	int* cuda_nodePtrs_h;
	int* cuda_nodeNeighbors_h;
	int* cuda_nodeVisited_h;
	int* cuda_currLevelNodes_h;
	int* cuda_nodeGate_h;
	int* cuda_nodeInput_h;
	int* cuda_nodeOutput_h;
	int* cuda_nextLevelNodes_h;
	int* cuda_numNextLevelNodes_h;

	//explicit memory allocation
	hipMalloc((void**)&cuda_nodePtrs_h, numNodePtrs*sizeof(int));
	hipMalloc((void**)&cuda_nodeNeighbors_h, numTotalNeighbors_h*sizeof(int));
	hipMalloc((void**)&cuda_nodeVisited_h, numNodes*sizeof(int));
	hipMalloc((void**)&cuda_currLevelNodes_h, numCurrLevelNodes*sizeof(int));
	hipMalloc((void**)&cuda_nodeGate_h, numNodes*sizeof(int));
	hipMalloc((void**)&cuda_nodeInput_h, numNodes*sizeof(int));
	hipMalloc((void**)&cuda_nodeOutput_h, numNodes*sizeof(int));
	hipMalloc((void**)&cuda_nextLevelNodes_h, sizeof(int) * numTotalNeighbors_h);
	hipMalloc((void**)&cuda_numNextLevelNodes_h, sizeof(int));
	/*cudaMallocManaged(&numNextLevelNodes_h, sizeof(int));
	int temp = 0;
	numNextLevelNodes_h = &temp;*/

	//copy to device
	hipMemcpy(cuda_nodePtrs_h, nodePtrs_h, numNodePtrs*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cuda_nodeNeighbors_h, nodeNeighbors_h, numTotalNeighbors_h * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cuda_nodeVisited_h, nodeVisited_h, numNodes * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cuda_currLevelNodes_h, currLevelNodes_h, numCurrLevelNodes * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cuda_nodeGate_h, nodeGate_h, numNodes * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cuda_nodeInput_h, nodeInput_h, numNodes * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cuda_nodeOutput_h, nodeOutput_h, numNodes * sizeof(int), hipMemcpyHostToDevice);
	//cudaMemset((void**)&cuda_numNextLevelNodes_h, 0, sizeof(int));

	//calculating the number of blocks needed, the grid, and block size
	int blocksize = 32;
	int  number_of_blocks = 10;


	global_queue <<<number_of_blocks, blocksize >>> (cuda_currLevelNodes_h, cuda_nodePtrs_h, cuda_nodeNeighbors_h, cuda_nodeVisited_h, cuda_nodeOutput_h, cuda_nodeGate_h, cuda_nodeInput_h, cuda_nextLevelNodes_h, cuda_numNextLevelNodes_h, numCurrLevelNodes);
	
	hipMemcpy(nextLevelNodes_h, cuda_nextLevelNodes_h, sizeof(int)*numTotalNeighbors_h, hipMemcpyDeviceToHost);
	hipMemcpy(nodeOutput_h, cuda_nodeOutput_h, sizeof(int)*numNodes, hipMemcpyDeviceToHost);
	hipMemcpy(numNextLevelNodes_h, cuda_numNextLevelNodes_h, sizeof(int), hipMemcpyDeviceToHost);

	fprintf(nextlevelnodes_file, "%d\n", *numNextLevelNodes_h);
	fprintf(nodeoutput_file, "%d\n", numNodes);
	printf("\n %d", numNextLevelNodes_h);
	
	for (int l = 0; l < *numNextLevelNodes_h; l++) {
		fprintf(nextlevelnodes_file, "%d\n", nextLevelNodes_h[l]);
	}
	for (int m = 0; m < numNodes; m++) {
		fprintf(nodeoutput_file, "%d\n", nodeOutput_h[m]);
	}
	fclose(nextlevelnodes_file);
	fclose(nodeoutput_file);
	return 0;
}
