#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "lodepng.h"
#include "gputimer.h"
#include <stdio.h>
#include <stdlib.h>

// rectify pixel values of a png image
__global__ void pooling(unsigned char* input, unsigned char* output, int width, int height, int n)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int row = index / width;
    int column = index / height;

    if (index < n && row%2==0 && column%2==0) {
        int png_index_1 = 4 * index;
        int png_index_2 = 4 * (index)+4;
        int png_index_3 = 4 * (index) + 4*width;
        int png_index_4 = 4 * (index) + 4*(width+1);
        //compare R
        int r = input[png_index_1];
        if (r < (int)input[png_index_2]) {
            r = input[png_index_2];
        }
        if (r < (int)input[png_index_3]) {
            r = input[png_index_3];
        }
        if (r < (int)input[png_index_4]) {
            r = input[png_index_4];
        }
        //compare G
        int g = input[png_index_1+1];
        if (g < (int)input[png_index_2+1]) {
            g = input[png_index_2+1];
        }
        if (g < (int)input[png_index_3+1]) {
            g = input[png_index_3+1];
        }
        if (g < (int)input[png_index_4+1]) {
            g = input[png_index_4+1];
        }
        //compare B
        int b = input[png_index_1+2];
        if (b < (int)input[png_index_2+2]) {
            b = input[png_index_2+2];
        }
        if (b < (int)input[png_index_3+2]) {
            b = input[png_index_3+2];
        }
        if (b < (int)input[png_index_4+2]) {
            b = input[png_index_4+2];
        }
        //compare A
        int a = input[png_index_1 + 3];
        if (a < (int)input[png_index_2 + 3]) {
            a = input[png_index_2 + 3];
        }
        if (a < (int)input[png_index_3 + 3]) {
            a = input[png_index_3 + 3];
        }
        if (a < (int)input[png_index_4 + 3]) {
            a= input[png_index_4 + 3];
        }
        //int new_image_index_row = (index % (width/2))/2;
        //int new_image_index = (index%width)/2 + height*(column/4);
        int new_image_index = (index % width) / 2 + (width / 2) * (row / 2);

        output[4 * new_image_index] = (unsigned char)r;
        output[4 * new_image_index + 1] = (unsigned char)g;
        output[4 * new_image_index + 2] = (unsigned char)b;
        output[4 * new_image_index + 3] = (unsigned char)a;
    }
      
}

int main(int argc, char** argv)
{

    char* png_input = "Test_1.png";
    char* png_output = "output.png";
    int thread_number = 32;

    unsigned error;
    
    unsigned char* image_host;
    unsigned char* image_cuda;
    unsigned char* new_image_cuda;
    unsigned width, height;
    unsigned* width_cuda;
    unsigned* height_cuda;

    

    error = lodepng_decode32_file(&image_host, &width, &height, png_input);
    if (error) printf("error %u: %s\n", error, lodepng_error_text(error));

    hipMalloc((void**)&image_cuda, width * height * 4 * sizeof(unsigned char));
    hipMalloc((void**)&new_image_cuda, width * height * sizeof(unsigned char));

    hipMemcpy(image_cuda, image_host, width * height * 4 * sizeof(unsigned char), hipMemcpyHostToDevice);

    //memcpy(image_cuda, image, width * height * 4 * sizeof(unsigned char));
    int  number_of_blocks = (width * height / thread_number);
    dim3 grid(number_of_blocks, 1, 1);
    dim3 block(thread_number, 1, 1);

    struct GpuTimer timer;
    timer.Start();

    pooling <<<grid, block>>> (image_cuda, new_image_cuda, width, height, height*width);
    timer.Stop();
    printf("timer: %f", timer.Elapsed());
    hipDeviceSynchronize();

    unsigned char* final_image = new unsigned char[width * height]();
    hipMemcpy(final_image, new_image_cuda, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);
    lodepng_encode32_file(png_output, final_image, width/2, height/2);
    hipFree(image_cuda);
    hipFree(new_image_cuda);

    return 0;
}
