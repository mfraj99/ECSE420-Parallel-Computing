#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "lodepng.h"
#include <stdio.h>
#include <stdlib.h>

// rectify pixel values of a png image
// input - pointer to array of pixels of input image
// output - pointer to array of pixels for output image
// n - limit of the numebr of threads
__global__ void rectification(unsigned char* input, unsigned char* output, int n, int width, int height)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n) {
        for (int j = 0; j < width; j++) {
            int png_index = 4 * width * index + 4 * j;

            if ((int)input[png_index] < 127) {
                output[png_index] = (unsigned char)127;
            }
            else {
                output[png_index] = input[png_index];
            }

            if ((int)input[png_index + 1] < 127) {
                output[png_index + 1] = (unsigned char)127;
            }
            else {
                output[png_index + 1] = input[png_index + 1];
            }

            if ((int)input[png_index + 2] < 127) {
                output[png_index + 2] = (unsigned char)127;
            }
            else {
                output[png_index + 2] = input[png_index + 2];
            }

            output[png_index + 3] = input[png_index + 3];
        }
    }
        
}

int main(int argc, char** argv)
{

    char* png_input = "Test_1.png";
    char* png_output = "output.png";
    int thread_number = 32;

    unsigned error;
    unsigned char* image = 0;
    unsigned width, height;

    error = lodepng_decode32_file(&image, &width, &height, png_input);
    if (error) printf("error %u: %s\n", error, lodepng_error_text(error));
    unsigned char* new_image = (unsigned char*)malloc(width * height * 4 * sizeof(unsigned char));
    unsigned char* final_image = (unsigned char*)malloc(width * height * 4 * sizeof(unsigned char));

    hipMallocManaged((void**)&image, width * height * 4 * sizeof(unsigned char));
    hipMallocManaged((void**)&new_image, width * height * 4 * sizeof(unsigned char));

    int threads_per_block = width*height/thread_number;

    rectification <<< (height+threads_per_block-1)/threads_per_block, threads_per_block >>> (image, new_image, height, width, height);

    hipDeviceSynchronize();

    //testing sequential version of rectification algorithm

    //unsigned char zero = (unsigned char)127;

    //for (int i = 0; i < height; i++) {
    //    for (int j = 0; j < width; j++) {	
    //    
    //        int index = 4 * i * width + 4 * j;
    //        printf("%d", (int)image[index]);
    //        printf("%u", (unsigned)image[index]);

    //        if ((int)image[index] < 127) {
    //            new_image[index] = zero;
    //        }
    //        else {
    //            new_image[index] = image[index];
    //        }

    //        if ((int)image[index + 1] < 127) {
    //            new_image[index + 1] = zero;
    //        }
    //        else {
    //            new_image[index + 1] = image[index + 1];
    //        }

    //        if ((int)image[index + 2] < 127) {
    //            new_image[index + 2] = zero;
    //        }
    //        else {
    //            new_image[index + 2] = image[index + 2];
    //        }

    //        new_image[index + 3] = image[index + 3];
    //    }
    //}

    //hipMemcpy(final_image, new_image, width * height * 4 * sizeof(unsigned char), hipMemcpyDeviceToHost);

    lodepng_encode32_file(png_output, new_image, width, height);

    hipFree(image);
    hipFree(new_image);

    /*free(image);
    free(new_image);*/

   /* if (strcmp(argv[0], "rectify") == 0) {
        char* png_input = argv[1];
        char* png_output = argv[2];
        int thread_number = atoi(argv[3]);

        

        return 0;
    }*/

    return 0;
}
